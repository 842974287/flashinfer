#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <flashinfer.cuh>
#include <type_traits>

#include "cpu_reference.h"
#include "utils.h"

template <typename T>
void _TestBatchDecodingKernelCorrectness(size_t page_size, size_t batch_size, size_t num_heads,
                                         size_t head_dim, flashinfer::RotaryMode rotary_mode) {
  std::vector<size_t> seq_lens(batch_size);
  utils::vec_randint_(seq_lens, 1, 256);
  std::vector<T> q;
  std::vector<T> o_ref;
  std::vector<T> kv_data;
  std::vector<size_t> kv_indptr{0};
  std::vector<size_t> kv_indices;
  std::vector<size_t> kv_last_page_offset;
  size_t page_counter = 0;

  for (size_t i = 0; i < batch_size; ++i) {
    size_t seq_len = seq_lens[i];
    size_t num_pages = (seq_len + page_size - 1) / page_size;
    size_t last_page_offset = (seq_len - 1) % page_size + 1;
    std::vector<T> qi(num_heads * head_dim), ki(num_heads * seq_len * head_dim),
        vi(num_heads * seq_len * head_dim);
    utils::vec_normal_(qi);
    utils::vec_normal_(ki);
    utils::vec_normal_(vi);

    // compute reference output
    std::vector<T> o_ref_i = cpu_reference::single_mha<T, T>(
        qi, ki, vi, num_heads, seq_len, head_dim, flashinfer::QKVLayout::kHND, rotary_mode);
    // append new q and o_ref
    q.insert(q.end(), qi.begin(), qi.end());
    o_ref.insert(o_ref.end(), o_ref_i.begin(), o_ref_i.end());
    // append new kv_indptr, kv_indices and kv_last_page_offset
    kv_last_page_offset.push_back(last_page_offset);
    kv_indptr.push_back(kv_indptr.back() + num_pages);
    for (size_t j = 0; j < num_pages; ++j) {
      kv_indices.push_back(page_counter++);
    }
    // append new pages to kv_data
    // step 1. first append (num_pages - 1) full pages
    for (size_t j = 0; j < num_pages - 1; ++j) {
      for (size_t h = 0; h < num_heads; ++h) {
        kv_data.insert(kv_data.end(),
                       ki.begin() + h * seq_len * head_dim + j * page_size * head_dim,
                       ki.begin() + h * seq_len * head_dim + (j + 1) * page_size * head_dim);
      }
      for (size_t h = 0; h < num_heads; ++h) {
        kv_data.insert(kv_data.end(),
                       vi.begin() + h * seq_len * head_dim + j * page_size * head_dim,
                       vi.begin() + h * seq_len * head_dim + (j + 1) * page_size * head_dim);
      }
    }
    // step 2. then append last page
    std::vector<T> padding((page_size - last_page_offset) * head_dim);
    for (size_t h = 0; h < num_heads; ++h) {
      kv_data.insert(kv_data.end(),
                     ki.begin() + h * seq_len * head_dim + (num_pages - 1) * page_size * head_dim,
                     ki.begin() + h * seq_len * head_dim + (num_pages - 1) * page_size * head_dim +
                         last_page_offset * head_dim);
      kv_data.insert(kv_data.end(), padding.begin(), padding.end());
    }
    for (size_t h = 0; h < num_heads; ++h) {
      kv_data.insert(kv_data.end(),
                     vi.begin() + h * seq_len * head_dim + (num_pages - 1) * page_size * head_dim,
                     vi.begin() + h * seq_len * head_dim + (num_pages - 1) * page_size * head_dim +
                         last_page_offset * head_dim);
      kv_data.insert(kv_data.end(), padding.begin(), padding.end());
    }
  }
  assert(kv_data.size() == page_counter * 1 * 2 * num_heads * page_size * head_dim);
  assert(q.size() == batch_size * num_heads * head_dim);
  assert(o_ref.size() == batch_size * num_heads * head_dim);

  // copy data to device
  thrust::device_vector<T> kv_data_device(kv_data);
  thrust::device_vector<size_t> kv_indptr_device(kv_indptr);
  thrust::device_vector<size_t> kv_indices_device(kv_indices);
  thrust::device_vector<size_t> kv_last_page_offset_device(kv_last_page_offset);
  thrust::device_vector<T> q_device(q);
  thrust::device_vector<T> o_device(o_ref.size());
  thrust::device_vector<float> tmp(8 * 1024 * 1024);

  // create paged_kv object
  flashinfer::paged_kv_t<T> paged_kv(page_counter, 1, 0, num_heads, page_size, head_dim, batch_size,
                                     thrust::raw_pointer_cast(kv_data_device.data()),
                                     thrust::raw_pointer_cast(kv_indptr_device.data()),
                                     thrust::raw_pointer_cast(kv_indices_device.data()),
                                     thrust::raw_pointer_cast(kv_last_page_offset_device.data()));

  // compute gpu result
  flashinfer::BatchDecodeWithPagedKVCache<T, T>(thrust::raw_pointer_cast(q_device.data()), paged_kv,
                                                thrust::raw_pointer_cast(o_device.data()),
                                                thrust::raw_pointer_cast(tmp.data()), batch_size,
                                                rotary_mode);

  // compare result
  thrust::host_vector<T> o_host = o_device;
  size_t num_result_errors_atol_1e_3_rtol_1e_3 = 0;
  bool nan_detected = false;
  for (size_t i = 0; i < batch_size * num_heads * head_dim; ++i) {
    if (std::isnan(float(o_host[i]))) {
      nan_detected = true;
    }
    num_result_errors_atol_1e_3_rtol_1e_3 +=
        (!utils::isclose(float(o_host[i]), float(o_ref[i]), 1e-3, 1e-3));
  }
  float result_accuracy =
      1. - float(num_result_errors_atol_1e_3_rtol_1e_3) / float(batch_size * num_heads * head_dim);
  std::cout << "page_size=" << page_size << ", num_heads=" << num_heads
            << ", batch_size=" << batch_size << ", head_dim=" << head_dim
            << ", rotary_mode=" << flashinfer::RotaryModeToString(rotary_mode)
            << ", result accuracy (atol=1e-3, rtol=1e-3): " << result_accuracy << std::endl;
  EXPECT_GT(result_accuracy, 0.90) << "Result correctness test failed.";
  EXPECT_EQ(nan_detected, false) << "NaN detected.";
}

template <typename T>
void TestBatchDecodeKernelCorrectness() {
  for (size_t page_size : {16}) {
    for (size_t batch_size : {1, 7, 37, 61}) {
      for (size_t num_heads : {32}) {
        for (size_t head_dim : {64, 128, 256}) {
          for (size_t rotary_mode : {0U, 1U}) {
            _TestBatchDecodingKernelCorrectness<T>(page_size, batch_size, num_heads, head_dim,
                                                   flashinfer::RotaryMode(rotary_mode));
          }
        }
      }
    }
  }
}

TEST(FlashInferCorrectnessTest, BatchDecodeKernelCorrectnessTestFP16) {
  TestBatchDecodeKernelCorrectness<half>();
}

TEST(FlashInferCorrectnessTest, TestBatchDecodeKernelCorrectnessBF16) {
  TestBatchDecodeKernelCorrectness<__hip_bfloat16>();
}

TEST(FlashInferCorrectnessTest, TestBatchDecodeKernelCorrectnessFP32) {
  TestBatchDecodeKernelCorrectness<float>();
}

TEST(FlashInferCorrectnessTest, TestBatchDecodeKernelCorrectnessE4M3) {
  TestBatchDecodeKernelCorrectness<__hip_fp8_e4m3_fnuz>();
}

TEST(FlashInferCorrectnessTest, TestBatchDecodeKernelCorrectnessE5M2) {
  TestBatchDecodeKernelCorrectness<__hip_fp8_e5m2_fnuz>();
}

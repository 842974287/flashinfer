#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <flashinfer/prefill.cuh>

#include "cpu_reference.h"
#include "utils.h"

using namespace flashinfer;

template <typename DTypeIn, typename DTypeOut>
void _TestSinglePrefillKernelCorrectness(size_t qo_len, size_t kv_len, size_t num_heads,
                                         size_t head_dim, QKVLayout layout,
                                         RotaryMode rotary_mode) {
  std::vector<DTypeIn> q(qo_len * num_heads * head_dim);
  std::vector<DTypeIn> k(kv_len * num_heads * head_dim);
  std::vector<DTypeIn> v(kv_len * num_heads * head_dim);
  std::vector<DTypeOut> o(qo_len * num_heads * head_dim);

  utils::vec_normal_(q);
  utils::vec_normal_(k);
  utils::vec_normal_(v);
  utils::vec_zero_(o);

  thrust::device_vector<DTypeIn> q_d(q);
  thrust::device_vector<DTypeIn> k_d(k);
  thrust::device_vector<DTypeIn> v_d(v);
  thrust::device_vector<DTypeOut> o_d(o);
  thrust::device_vector<float> tmp_d(kv_len * num_heads * head_dim);

  hipError_t status = flashinfer::SinglePrefillWithKVCache<DTypeIn, DTypeOut>(
      thrust::raw_pointer_cast(q_d.data()), thrust::raw_pointer_cast(k_d.data()),
      thrust::raw_pointer_cast(v_d.data()), thrust::raw_pointer_cast(o_d.data()),
      thrust::raw_pointer_cast(tmp_d.data()), num_heads, qo_len, kv_len, head_dim, layout,
      rotary_mode);

  thrust::host_vector<DTypeOut> o_h(o_d);
  std::vector<DTypeOut> o_ref = cpu_reference::single_mha<DTypeIn, DTypeOut>(
      q, k, v, qo_len, kv_len, num_heads, head_dim, layout, rotary_mode);
  size_t num_results_error_atol_1e_3_rtol_1e_3 = 0;
  bool nan_detected = false;

  for (size_t i = 0; i < o_ref.size(); ++i) {
    if (isnan(float(o_h[i]))) {
      nan_detected = true;
    }
    num_results_error_atol_1e_3_rtol_1e_3 +=
        (!utils::isclose(float(o_ref[i]), float(o_h[i]), 1e-3, 1e-3));
    if (!utils::isclose(float(o_ref[i]), float(o_h[i]), 1e-3, 1e-3)) {
      std::cout << "row=" << i / head_dim << ", rol=" << i % head_dim << ", ref=" << float(o_ref[i])
                << ", result=" << float(o_h[i]) << std::endl;
    }
  }

  float result_accuracy = 1. - float(num_results_error_atol_1e_3_rtol_1e_3) / float(o_ref.size());
  std::cout << "num_heads=" << num_heads << ", qo_len=" << qo_len << ", kv_len=" << kv_len
            << ", head_dim=" << head_dim << ", layout=" << QKVLayoutToString(layout)
            << ", rotary_mode=" << RotaryModeToString(rotary_mode)
            << ", result_accuracy=" << result_accuracy << std::endl;
  EXPECT_GT(result_accuracy, 0.90) << "Result correctness test failed.";
  EXPECT_FALSE(nan_detected) << "Nan detected in the result.";
}

template <typename DTypeIn, typename DTypeOut>
void TestSinglePrefillKernelCorrectness() {
  for (size_t qo_len : {399, 400, 401}) {
    for (size_t kv_len : {533, 534, 535}) {
      for (size_t num_heads : {32}) {
        for (size_t head_dim : {64, 128}) {
          for (size_t rotary_mode : {0, 1}) {
            for (size_t layout : {0, 1}) {
              _TestSinglePrefillKernelCorrectness<DTypeIn, DTypeOut>(
                  qo_len, kv_len, num_heads, head_dim, QKVLayout(layout), RotaryMode(rotary_mode));
            }
          }
        }
      }
    }
  }
}

TEST(FlashInferCorrectnessTest, SinglePrefillKernelCorrectnessTestFP16) {
  TestSinglePrefillKernelCorrectness<half, half>();
}

TEST(FlashInferCorrectnessTest, SinglePrefillKernelCorrectnessTestBF16) {
  TestSinglePrefillKernelCorrectness<hip_bfloat16, hip_bfloat16>();
}
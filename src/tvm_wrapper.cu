#include "hip/hip_runtime.h"
#include <dlpack/dlpack.h>
#include <tvm/runtime/logging.h>
#include <tvm/runtime/module.h>
#include <tvm/runtime/ndarray.h>
#include <tvm/runtime/packed_func.h>
#include <tvm/runtime/registry.h>

#include <flashinfer.cuh>

using tvm::runtime::DataType;
using tvm::runtime::NDArray;
using tvm::runtime::ShapeTuple;

#define SWITCH_TVM_CUDA_DTYPE(dl_dtype, cuda_dtype, ...)          \
  if (dl_dtype.code == kDLFloat && dl_dtype.bits == 16) {         \
    using cuda_dtype = half;                                      \
    __VA_ARGS__                                                   \
  } else if (dl_dtype.code == kDLFloat && dl_dtype.bits == 32) {  \
    using cuda_dtype = float;                                     \
    __VA_ARGS__                                                   \
  } else if (dl_dtype.code == kDLBfloat && dl_dtype.bits == 16) { \
    using cuda_dtype = hip_bfloat16;                               \
    __VA_ARGS__                                                   \
  } else if (dl_dtype.code == DataType::kE4M3Float) {             \
    using cuda_dtype = __hip_fp8_e4m3_fnuz;                             \
    __VA_ARGS__                                                   \
  } else if (dl_dtype.code == DataType::kE5M2Float) {             \
    using cuda_dtype = __hip_fp8_e5m2_fnuz;                             \
    __VA_ARGS__                                                   \
  } else {                                                        \
    LOG(FATAL) << "Unsupported data type " << dl_dtype.code;      \
  }

int _FlashInferSingleDecodeWithKVCache(DLTensor *q, DLTensor *k, DLTensor *v, DLTensor *tmp,
                                       int64_t rotary_mode, double rope_scale, double rope_theta,
                                       DLTensor *o) {
  CHECK_EQ(q->device.device_type, kDLCUDA) << "The device of q matrix must be CUDA.";
  CHECK_EQ(k->device.device_type, kDLCUDA) << "The device of k matrix must be CUDA.";
  CHECK_EQ(v->device.device_type, kDLCUDA) << "The device of v matrix must be CUDA.";
  CHECK_EQ(tmp->device.device_type, kDLCUDA) << "The device of tmp matrix must be CUDA.";
  CHECK_EQ(o->device.device_type, kDLCUDA) << "The device of o matrix must be CUDA.";

  size_t dev_id = q->device.device_id;
  CHECK_EQ(k->device.device_id, dev_id) << "The device id of q and k matrix doesn't match.";
  CHECK_EQ(v->device.device_id, dev_id) << "The device id of q and v matrix doesn't match.";
  CHECK_EQ(tmp->device.device_id, dev_id) << "The device id of q and tmp matrix doesn't match.";
  CHECK_EQ(o->device.device_id, dev_id) << "The device id of q and o matrix doesn't match.";

  CHECK_EQ(q->ndim, 2);
  size_t num_heads = q->shape[0];
  size_t head_dim = q->shape[1];
  CHECK_EQ(k->ndim, 3);
  size_t seq_len = k->shape[0];
  CHECK_EQ(k->shape[1], num_heads);
  CHECK_EQ(k->shape[2], head_dim);
  CHECK_EQ(v->ndim, 3);
  CHECK_EQ(v->shape[0], seq_len);
  CHECK_EQ(v->shape[1], num_heads);
  CHECK_EQ(v->shape[2], head_dim);
  CHECK_EQ(o->ndim, 2);
  CHECK_EQ(o->shape[0], num_heads);
  CHECK_EQ(o->shape[1], head_dim);

  CHECK(q->dtype.lanes == 1 && k->dtype.lanes == 1 && v->dtype.lanes == 1);
  CHECK(q->dtype.bits == k->dtype.bits && q->dtype.code == k->dtype.code);
  CHECK(q->dtype.bits == v->dtype.bits && q->dtype.code == v->dtype.code);

  SWITCH_TVM_CUDA_DTYPE(
      q->dtype, dtype_in, {SWITCH_TVM_CUDA_DTYPE(o->dtype, dtype_out, {
        hipError_t status = flashinfer::SingleDecodeWithKVCache(
            (dtype_in *)q->data, (dtype_in *)k->data, (dtype_in *)v->data, (dtype_out *)o->data,
            (float *)tmp->data, num_heads, seq_len, head_dim, flashinfer::RotaryMode(rotary_mode),
            rope_scale, rope_theta, 0, dev_id);
        if (status != hipSuccess) {
          LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
        }
      })});
  return 0;
}

TVM_DLL_EXPORT_TYPED_FUNC(FlashInferSingleDecodeWithKVCache, _FlashInferSingleDecodeWithKVCache);
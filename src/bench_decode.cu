#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>

#include <flashinfer.cuh>
#include <nvbench/nvbench.cuh>

using flashinfer::QKVLayout;
using flashinfer::RotaryMode;

template <typename dtype_in, typename dtype_out, size_t rotary_mode, size_t layout>
void bench_flashinfer_decode(nvbench::state &state) {
  size_t seq_len = state.get_int64("seq_len");
  size_t num_heads = state.get_int64("num_heads");
  size_t head_dim = state.get_int64("head_dim");
  // Allocate input data:
  thrust::device_vector<dtype_in> Q(num_heads * head_dim);
  thrust::device_vector<dtype_in> K(seq_len * num_heads * head_dim);
  thrust::device_vector<dtype_in> V(seq_len * num_heads * head_dim);
  thrust::device_vector<dtype_out> O(num_heads * head_dim);
  thrust::device_vector<float> tmp(512 * num_heads * head_dim);

  // Provide throughput information:
  state.add_global_memory_reads<dtype_in>(num_heads * head_dim + 2 * seq_len * num_heads * head_dim,
                                          "Read");
  state.add_global_memory_writes<dtype_out>(num_heads * head_dim, "Write");

  state.exec(nvbench::exec_tag::timer, [&](nvbench::launch &launch, auto &timer) {
    timer.start();
    hipError_t status = flashinfer::SingleDecodeWithKVCache(
        thrust::raw_pointer_cast(Q.data()), thrust::raw_pointer_cast(K.data()),
        thrust::raw_pointer_cast(V.data()), thrust::raw_pointer_cast(O.data()),
        thrust::raw_pointer_cast(tmp.data()), num_heads, seq_len, head_dim, QKVLayout(layout),
        RotaryMode(rotary_mode), 1.f, 1e4, launch.get_stream());
    if (status != hipSuccess) {
      state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
    }
    timer.stop();
  });
}

#define STR_HELPER(x) #x
#define STR(x) STR_HELPER(x)
#define BENCH_FLASHINFER_DECODE(dtype_in, dtype_out, rotary_mode, layout)                       \
  auto bench_flashinfer_decode_##dtype_in##_##dtype_out##_##rotary_mode##_##layout##_ =         \
      bench_flashinfer_decode<dtype_in, dtype_out, rotary_mode, layout>;                        \
  NVBENCH_BENCH(bench_flashinfer_decode_##dtype_in##_##dtype_out##_##rotary_mode##_##layout##_) \
      .set_name(("bench_flashinfer_" STR(dtype_in) "_" STR(dtype_out) "_") +                    \
                flashinfer::RotaryModeToString(RotaryMode(rotary_mode)) + "_" +                 \
                flashinfer::QKVLayoutToString(QKVLayout(layout)))                               \
      .add_int64_axis("seq_len", {32, 64, 128, 256, 512, 1024, 2048, 4096, 8192, 16384, 32768}) \
      .add_int64_axis("num_heads", {32})                                                        \
      .add_int64_axis("head_dim", {128})

BENCH_FLASHINFER_DECODE(half, half, 0U, 0U);
BENCH_FLASHINFER_DECODE(half, half, 0U, 1U);
BENCH_FLASHINFER_DECODE(half, half, 1U, 0U);
BENCH_FLASHINFER_DECODE(half, half, 1U, 1U);
BENCH_FLASHINFER_DECODE(__hip_fp8_e5m2_fnuz, half, 0U, 0U);
BENCH_FLASHINFER_DECODE(__hip_fp8_e5m2_fnuz, half, 0U, 1U);
BENCH_FLASHINFER_DECODE(__hip_fp8_e5m2_fnuz, half, 1U, 0U);
BENCH_FLASHINFER_DECODE(__hip_fp8_e5m2_fnuz, half, 1U, 1U);
